#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <random>
#include <limits>

// --- DATA --- // 

// Character frequencies (using a fixed array for GPU compatibility)
__constant__ double frequencyArray[26] = {
    7.03, 1.41, 1.83, 3.23, 11.97, 2.00, 1.94, 5.08, 5.92, 0.11, 
    0.90, 4.07, 2.29, 6.54, 8.38, 1.47, 0.04, 5.21, 5.73, 8.62, 
    3.29, 1.27, 2.19, 0.10, 2.63, 0.05
};

// GPU-friendly key structure
struct Key {
    int x;
    int y;
    int finger;
    int hand;
};

// CUDA Error Checking Macro
#define CUDA_CHECK(call) \
    if ((call) != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(hipGetLastError()) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    }

// --- GPU Kernels --- //

// Kernel to initialize curand states for randomness
__global__ void initCurandStates(hiprandState *state, int seed, int populationSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < populationSize) {
        hiprand_init(seed + idx, 0, 0, &state[idx]);
    }
}

// Kernel for fitness evaluation
__global__ void evaluateFitness(
    const char *layouts, 
    const int *keys, 
    double *fitness, 
    int populationSize, 
    int layoutSize
) {
    __shared__ Key sharedKeys[40];
    if (threadIdx.x < 40) {
        sharedKeys[threadIdx.x] = reinterpret_cast<const Key*>(keys)[threadIdx.x];
    }
    __syncthreads();

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < populationSize) {
        double value = 0.0;
        for (int i = 0; i < layoutSize; ++i) {
            char c = layouts[idx * layoutSize + i];
            double freq = frequencyArray[c - 'a'];

            const Key &key = sharedKeys[i];
            if (key.finger == 0 || key.finger == 9) value -= freq * 2;
            if (key.y == 1) value += freq * 1.5;
        }
        fitness[idx] = value;
    }
}

// Kernel for mutation
__global__ void mutateKernel(char *population, hiprandState *state, int layoutSize, float mutationRate, int populationSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < populationSize) {
        hiprandState localState = state[idx];
        for (int i = 0; i < layoutSize; ++i) {
            if (hiprand_uniform(&localState) < mutationRate) {
                int swapIdx = hiprand(&localState) % layoutSize;
                char temp = population[idx * layoutSize + i];
                population[idx * layoutSize + i] = population[idx * layoutSize + swapIdx];
                population[idx * layoutSize + swapIdx] = temp;
            }
        }
        state[idx] = localState;
    }
}

// Kernel for crossover
__global__ void crossoverKernel(
    const char *parents, 
    char *offspring, 
    hiprandState *state, 
    int layoutSize, 
    int populationSize
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < populationSize / 2) {
        hiprandState localState = state[idx];
        int parent1Idx = hiprand(&localState) % populationSize;
        int parent2Idx = hiprand(&localState) % populationSize;

        for (int i = 0; i < layoutSize / 2; ++i) {
            offspring[idx * layoutSize + i] = parents[parent1Idx * layoutSize + i];
        }
        for (int i = layoutSize / 2; i < layoutSize; ++i) {
            offspring[idx * layoutSize + i] = parents[parent2Idx * layoutSize + i];
        }
        state[idx] = localState;
    }
}

// --- GPU Genetic Algorithm --- //

void runGeneticAlgorithmGPU(int populationSize, int generations, float mutationRate, const std::string &characters) {
    int layoutSize = characters.size();
    int numKeys = 40; // Assuming 10 keys per row, 4 rows

    // Initialize keyboard
    std::vector<Key> keys(numKeys);
    for (int i = 0; i < numKeys; ++i) {
        keys[i] = {i % 10, i / 10, i % 10, i / 5};
    }

    // Initialize population
    std::vector<char> flatPopulation(populationSize * layoutSize);
    std::generate(flatPopulation.begin(), flatPopulation.end(), [&characters]() {
        return characters[std::rand() % characters.size()];
    });

    // Allocate GPU memory
    char *d_population;
    char *d_offspring;
    double *d_fitness;
    int *d_keys;
    hiprandState *d_states;

    CUDA_CHECK(hipMalloc(&d_population, populationSize * layoutSize * sizeof(char)));
    CUDA_CHECK(hipMalloc(&d_offspring, (populationSize / 2) * layoutSize * sizeof(char)));
    CUDA_CHECK(hipMalloc(&d_fitness, populationSize * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_keys, numKeys * sizeof(Key)));
    CUDA_CHECK(hipMalloc(&d_states, populationSize * sizeof(hiprandState)));

    // Copy data to GPU
    CUDA_CHECK(hipMemcpy(d_population, flatPopulation.data(), populationSize * layoutSize * sizeof(char), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_keys, keys.data(), numKeys * sizeof(Key), hipMemcpyHostToDevice));

    // Initialize random states
    initCurandStates<<<(populationSize + 255) / 256, 256>>>(d_states, time(nullptr), populationSize);

    // Configure kernel dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (populationSize + threadsPerBlock - 1) / threadsPerBlock;

    // Run genetic algorithm
    for (int gen = 0; gen < generations; ++gen) {
        evaluateFitness<<<blocksPerGrid, threadsPerBlock>>>(d_population, d_keys, d_fitness, populationSize, layoutSize);

        // Perform selection, crossover, and mutation
        crossoverKernel<<<blocksPerGrid, threadsPerBlock>>>(d_population, d_offspring, d_states, layoutSize, populationSize);
        mutateKernel<<<blocksPerGrid, threadsPerBlock>>>(d_offspring, d_states, layoutSize, mutationRate, populationSize / 2);

        // Replace population with offspring
        CUDA_CHECK(hipMemcpy(d_population, d_offspring, (populationSize / 2) * layoutSize * sizeof(char), hipMemcpyDeviceToDevice));
    }

    // Retrieve results
    std::vector<char> finalPopulation(populationSize * layoutSize);
    CUDA_CHECK(hipMemcpy(finalPopulation.data(), d_population, populationSize * layoutSize * sizeof(char), hipMemcpyDeviceToHost));

    hipFree(d_population);
    hipFree(d_offspring);
    hipFree(d_fitness);
    hipFree(d_keys);
    hipFree(d_states);
}

// --- Main Function --- //

int main() {
    const std::string characters = "abcdefghijklmnopqrstuvwxyz";
    const int populationSize = 1000;
    const int generations = 1000;
    const float mutationRate = 0.05;

    runGeneticAlgorithmGPU(populationSize, generations, mutationRate, characters);

    return 0;
}
